// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -out-root %T/fence %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/fence/fence.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/fence/fence.dp.cpp -o %T/fence/fence.dp.o %}

// clang-format off
// CHECK: #include <cmath>
#include <hip/hip_runtime.h>
#include <cstdint>

__global__ void fence() {

  // CHECK: sycl::group_barrier(item_ct1.get_group());
  asm("fence.acq_rel.gpu;");
  
}

// clang-format on
